#include "hip/hip_runtime.h"
#include "lsystem.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>

lsystem::lsystem(std::string axiom, std::map<char, std::string> rules): axiom(""), rules({})
{
   this->axiom = axiom;
   this->rules = rules;
}

lsystem::lsystem(std::string axiom, std::string rules): axiom(""), rules({})
{
    this->axiom = axiom;

    try
    {
        parseString(rules);
    }
    catch(const RulesWrongFormatException &e)
    {
        std::cout << "Exception occured: " << e.what() << std::endl;
    }
    
}

lsystem::lsystem(const lsystem& other)
{
    this->axiom = other.axiom;
    this->rules = other.rules;
}

lsystem& lsystem::operator=(const lsystem &other)
{
    if(this != &other)
    {
        lsystem tmp(other);
        std::swap(tmp.axiom, this->axiom);
        std::swap(tmp.rules, this->rules);
    }

    return *this;
}

lsystem::~lsystem()
{
    if(this->rulesKey != nullptr)
    {
       hipFree(this->rulesKey);
       hipFree(this->rulesValueLength);

       for (int i = 0; i < this->rulesLength; i++)
       {
            hipFree(this->rulesValue[i]); 
       }
       hipFree(this->rulesValue);
    }
}

std::string lsystem::get_axiom() const
{
    return this->axiom;
}

std::map<char, std::string> lsystem::get_rules() const
{
    return this->rules;
}

std::string lsystem::get_result() const
{
    return this->result;
}

std::string lsystem::get_GPUResult() const
{
    return this->GPUresult;
}

std::ostream &operator<<(std::ostream &os, const lsystem &system)
{
    if(system.rules.size() == 0 || system.axiom.empty())
    {
        throw RulesEmptyException();
    }

    os << "l-system with axiom: " << system.get_axiom() << " and rules: ";

    for (const auto& [key, value] : system.get_rules())
    {
        os << key << " → " << value << "    ";
    }

    return os;
}

void lsystem::parseString(const std::string rules)
{
    std::stringstream ss(rules);
    char key;
    std::string value;

    while(ss >> key)
    {
        if(!(ss >> value))
        {
            this->rules = {};
            this->axiom = "";
            throw RulesWrongFormatException();
        }

        // std::cout << "Key: " << key << " value: " << value << std::endl;

        this->rules[key] = value;
    }
}

void lsystem::setCustomMeaning(std::string symbols, int meaning)
{
    for(char c: symbols)
    {
        if(c != ' ')
        {
            this->meanings[c] = meaning;
        }
    }
}

void lsystem::execute(const int iteration)
{
    std::string result = this->axiom;

    for(int i = 0; i < iteration; i++)
    {
        std::string tmp = "";

        for(char c: result)
        {
            if(this->rules.find(c) != this->rules.end())
            {
                tmp += this->rules[c];
            }
            else
            {
                tmp += c;
            }
        }

        result = tmp;
    }

    // std::cout << "The result is: " << result << std::endl;
    this->result = result;
}

void lsystem::write(std::string name) const
{
    std::ofstream file("../Results/" + name + ".txt", std::ios::out);
    
    if (file.is_open())
    {
        file << this->result;
        file.close();

        std::cout << "The string has been written to the file." << std::endl;
    }
    else
    {
        std::cout << "Unable to open file." << std::endl;
    }
}

void lsystem::draw(const std::string name, const double turnAngle, const int stepLength, const int startingDirection)
{
    if(this->meanings.size() == 0)
    {
        this->meanings = this->defaultMeaning;
    }

    std::ofstream file("../Results/" + name + ".svg");
    if (!file.is_open()) {
        std::cout << "Errore nell'apertura del file." << std::endl;
        return;
    }

    file << "<svg xmlns=\"http://www.w3.org/2000/svg\" version=\"1.1\">" << std::endl;

    double x = 0.0;
    double y = 0.0;

    double angle = startingDirection;

    for (char c : this->result) {
        if (this->meanings[c] == DRAW) {
            
            double newX = x + stepLength * cos( angle * M_PI / 180.0 );
            double newY = y + stepLength * sin( angle * M_PI / 180.0 );

            file << "<line x1=\"" << static_cast<int>(x) << "\" y1=\"" << static_cast<int>(y)<< "\" x2=\"" << static_cast<int>(newX) << "\" y2=\"" << static_cast<int>(newY) << "\" stroke=\"black\" stroke-width=\"0.8\"/>" << std::endl;

            x = newX;
            y = newY;
        }
        else if (this->meanings[c] == TURNRIGHT) {
            angle += turnAngle;
        }
        else if (this->meanings[c] == TURNLEFT) {
            angle -= turnAngle;
        }
        else if(this->meanings[c] == MOVE)
        {
            double newX = x + stepLength * cos( angle * M_PI / 180.0 );
            double newY = y + stepLength * sin( angle * M_PI / 180.0 );

            x = newX;
            y = newY;
        }
        else if (this->meanings[c] == PUSH)
        {
            this->states.push(angle);
            this->states.push(y);
            this->states.push(x);
        }
        else if (this->meanings[c] == POP)
        {
            x = this->states.top();
            this->states.pop();

            y = this->states.top();
            this->states.pop();

            angle = this->states.top();
            this->states.pop();
        }
    }

    file << "</svg>" << std::endl;
    file.close();

    std::cout << "Image successfully generated: " << name << std::endl;
}

/*************************************************
*                                                *
*                   GPU STUFF                    *
*                                                *
**************************************************/

__global__ void countKernel(const char* axiom, int* out, const char* rulesKey, const int* rulesValueLength, int axiomLength, int rulesLength)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    bool found = false;

    if(tid < axiomLength)
    {
        for (int i = 0; i < rulesLength; i++)
        {
            if(axiom[tid] == rulesKey[i])
            {
                out[tid] += rulesValueLength[i];
                found = true;
                break; 
            }
        }

        if(!found)
        {
            out[tid] += 1;
        }
    }
}

__global__ void RewritingKernel(char* input, char* out, char* rulesKey, int* rulesValueLength, char** rulesValue, int* offsetArray, int inputLength, int rulesLength)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    bool found = false;

    if(tid < inputLength)
    {
        for (int i = 0; i < rulesLength; ++i)
        {
            if(input[tid] == rulesKey[i])
            {
                // printf("Input[%d]: %c\tLetter[%d]: %c\n", tid, input[tid], i, rulesKey[i]);

                found = true;

                int size = rulesValueLength[i];
                for (int j = 0; j < size; j++)
                {
                    // printf("out[%d] = parameters[%d][%d] -> %c\n", offsetArray[tid] + j, i, j, rulesValue[i][j]);
                    out[offsetArray[tid] + j] = rulesValue[i][j];
                }

                break;
            }
        }

        if(!found)
        {
            // printf("out[%d] = input[%d] -> %c\n", offsetArray[tid], tid, input[tid]);
            out[offsetArray[tid]] = input[tid]; 
        }
    }
}

void PrintArray(int* array, int length)
{
    for (int i = 0; i < length; i++)
    {
        std::cout << array[i] << " ";
    }
    std::cout << std::endl;
    
}

void lsystem::execute(const int iteration, const bool useGPU)
{
    setupGPUstuff();

    for (int i = 0; i < iteration; i++)
    {
        // std::cout << "Input: " << this->GPUresult << std::endl;
 
        int* valuesLength = count();
        // std::cout << "Counting substitute string length: ";
        // PrintArray(valuesLength, this->GPUresult.length());

        int* offsetArray;
        hipMallocManaged(&offsetArray, (GPUresult.length() + 1) * sizeof(int)); 
        offsetArray = prefixSum(valuesLength);

        // std::cout << "Calculating offset array: ";
        // PrintArray(offsetArray, this->GPUresult.length() + 1);

        rewrite(offsetArray);
        // std::cout << "Output: " << this->GPUresult << std::endl;
        
        hipFree(valuesLength);
        hipFree(offsetArray);
    }
}

void lsystem::setupGPUstuff()
{
    this->GPUresult = this->axiom;

    this->rulesLength = this->rules.size();
    hipMallocManaged(&this->rulesKey, this->rulesLength * sizeof(char));
    hipMallocManaged(&this->rulesValueLength, this->rulesLength * sizeof(int));

    // Allocating memory for rules string on GPU
    hipMallocManaged(&this->rulesValue, this->rulesLength * sizeof(char*));    

    int i = 0;
    for (const auto& [key, value] : this->rules)
    {
        this->rulesKey[i] = key;
        this->rulesValueLength[i] = value.length();

        // Allocating on GPU memory arrays of string
        hipMallocManaged(&this->rulesValue[i], (value.length() + 1) * sizeof(char));
        strcpy(this->rulesValue[i], value.c_str());

        i++;
    }
}

int* lsystem::count()
{
    int threads = 1024;
    int blocks = (this->GPUresult.length() + threads - 1) / threads;

    int* out;
    hipMallocManaged(&out, this->GPUresult.length() * sizeof(int));
    hipMemset(out, 0, this->GPUresult.length() * sizeof(int));

    char* axiom;
    hipMallocManaged(&axiom, this->GPUresult.length() * sizeof(char));
    memcpy(axiom, this->GPUresult.c_str(), this->GPUresult.length() * sizeof(char));

    countKernel<<<blocks, threads>>>(axiom, out, this->rulesKey, this->rulesValueLength, this->GPUresult.length(), this->rulesLength);
    hipDeviceSynchronize();

    return out;
}

int* lsystem::prefixSum(int* input)
{
    thrust::device_vector<int> d_input(input, input + this->GPUresult.length());
    thrust::device_vector<int> d_output(this->GPUresult.length() + 1);

    thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin() + 1);

    int* result;
    hipMallocManaged(&result, (this->GPUresult.length() + 1) * sizeof(int));
    hipMemcpy(result, thrust::raw_pointer_cast(d_output.data()), (this->GPUresult.length() + 1) * sizeof(int), hipMemcpyDeviceToDevice);

    return result;
}

void lsystem::rewrite(int* offsetArray)
{
    char* input;
    hipMallocManaged(&input, this->GPUresult.length() * sizeof(char));
    memcpy(input, this->GPUresult.c_str(), this->GPUresult.length() * sizeof(char));

    char* out;
    hipMallocManaged(&out, offsetArray[this->GPUresult.length()] * sizeof(char));
    hipMemset(out, 0, offsetArray[this->GPUresult.length()] * sizeof(char));

    int threads = 1024;
    int blocks = (this->GPUresult.length() * threads - 1) / threads;

    RewritingKernel<<<blocks, threads>>>(input, out, this->rulesKey, this->rulesValueLength, this->rulesValue, offsetArray, this->GPUresult.length(), this->rulesLength);
    hipDeviceSynchronize();

    this->GPUresult = out;

    hipFree(input);
    hipFree(out);
}
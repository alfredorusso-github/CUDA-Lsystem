#include "hip/hip_runtime.h"
#include "lsystem.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>

lsystem::lsystem(std::string axiom, std::string rules): axiom(""), rules({})
{
    this->axiom = axiom;

    if(axiom.length() == 0)
    {
        throw AxiomEmptyExecption();
    }

    if(rules.length() == 0)
    {
        throw RulesEmptyException();
    }

    try
    {
        parseString(rules);
    }
    catch(const RulesWrongFormatException &e)
    {
        std::cout << "Exception occured: " << e.what() << std::endl;
    }
    
}

lsystem::lsystem(std::string axiom, std::map<char, std::string> rules): axiom(""), rules({})
{
    if(axiom.length() == 0)
    {
        throw AxiomEmptyExecption();
    } 

    if(rules.empty())
    {
        throw RulesEmptyException();
    }  

    this->axiom = axiom;
    this->rules = rules;
}

lsystem::lsystem(const lsystem& other)
{
    this->axiom = other.axiom;
    this->rules = other.rules;
}

lsystem& lsystem::operator=(const lsystem &other)
{
    if(this != &other)
    {
        lsystem tmp(other);
        std::swap(tmp.axiom, this->axiom);
        std::swap(tmp.rules, this->rules);
    }

    return *this;
}

lsystem::~lsystem()
{
    std::cout << "Calling destructor" << std::endl;
    freeMemory();
}

void lsystem::freeMemory()
{
    if(this->isGpuUsed && this->isFreeNeeded)
    {
        hipFree(this->rulesKey);
        hipFree(this->rulesValueLength);

        for (int i = 0; i < this->rulesLength; i++)
        {
            hipFree(this->rulesValue[i]); 
        }
        hipFree(this->rulesValue);

        isFreeNeeded = false;
        isGpuUsed = false;
    }
}

std::string lsystem::get_axiom() const
{
    return this->axiom;
}

std::map<char, std::string> lsystem::get_rules() const
{
    return this->rules;
}

std::string lsystem::get_result() const
{
    if(this->result.length() == 0)
    {
        throw EmptyResultExecption();
    }

    return this->result;
}

std::string lsystem::get_GPUResult() const
{
    if(this->GPUresult.length() == 0)
    {
        throw EmptyGpuResultExecption();
    }

    return this->GPUresult;
}

std::map<char, int> lsystem::get_meanings() const
{
    return this->meanings;
}

std::ostream &operator<<(std::ostream &os, const lsystem &system)
{
    if(system.rules.size() == 0 || system.axiom.empty())
    {
        throw RulesEmptyException();
    }

    os << "l-system specs" << std::endl;
    os << "axiom: " << system.get_axiom() << std::endl;

    os << "rules: "; 
    for (const auto& [key, value] : system.get_rules())
    {
        os << key << " → " << value << "  ";
    }

    os << std::endl <<  "meanings: ";
    for (const auto& [key, value] : system.get_meanings())
    {
        os << key << " → " << system.symbolMeaningsName[value] << "  ";
    }

    return os;
}

void lsystem::parseString(const std::string rules)
{
    std::stringstream ss(rules);
    char key;
    std::string value;

    while(ss >> key)
    {
        if(!(ss >> value))
        {
            this->rules = {};
            this->axiom = "";
            throw RulesWrongFormatException();
        }

        // std::cout << "Key: " << key << " value: " << value << std::endl;

        this->rules[key] = value;
    }
}

void lsystem::setMeaning(std::string symbols, int meaning)
{
    for(char c: symbols)
    {
        if(c != ' ')
        {
            this->meanings[c] = meaning;
        }
    }
}

void lsystem::execute(const int iteration)
{
    std::string result = this->axiom;

    for(int i = 0; i < iteration; i++)
    {
        std::string tmp = "";

        for(char c: result)
        {
            if(this->rules.find(c) != this->rules.end())
            {
                tmp += this->rules[c];
            }
            else
            {
                tmp += c;
            }
        }

        result = tmp;
    }

    // std::cout << "The result is: " << result << std::endl;
    this->result = result;
}

void lsystem::write(const std::string name) const
{
    if(this->result.length() == 0)
    {
        throw EmptyResultExecption();
    }

    if(this->GPUresult.length() == 0)
    {
        throw EmptyGpuResultExecption();
    }

    std::ofstream file("../Results/" + name + ".txt", std::ios::out);
    
    if (file.is_open())
    {
        file << this->result;
        file.close();

        std::cout << "The string has been written to the file." << std::endl;
    }
    else
    {
        std::cout << "Unable to open file." << std::endl;
    }
}

void lsystem::draw(const std::string name, const double turnAngle, const int stepLength, const bool drawGPUResult, const int startingDirection)
{
    if(drawGPUResult && this->GPUresult.length() == 0)
    {
        throw EmptyGpuResultExecption();
    }

    if(this->result.length() == 0)
    {
        throw EmptyResultExecption();
    }

    std::ofstream file("../Results/" + name + ".svg");
    if (!file.is_open()) {
        std::cerr << "An error occured when tried to open/create the files." << std::endl;
        return;
    }

    file << "<svg xmlns=\"http://www.w3.org/2000/svg\" version=\"1.1\">" << std::endl;

    double x = 0.0;
    double y = 0.0;

    double angle = startingDirection;

    std::string whatToDraw = drawGPUResult ? this->GPUresult : this->result;

    for (char c : whatToDraw) {
        if (this->meanings[c] == Draw) {
            
            double newX = x + stepLength * cos( angle * M_PI / 180.0 );
            double newY = y + stepLength * sin( angle * M_PI / 180.0 );

            file << "<line x1=\"" << static_cast<int>(x) << "\" y1=\"" << static_cast<int>(y)<< "\" x2=\"" << static_cast<int>(newX) << "\" y2=\"" << static_cast<int>(newY) << "\" stroke=\"black\" stroke-width=\"0.8\"/>" << std::endl;

            x = newX;
            y = newY;
        }
        else if (this->meanings[c] == Turnright) {
            angle += turnAngle;
        }
        else if (this->meanings[c] == Turnleft) {
            angle -= turnAngle;
        }
        else if(this->meanings[c] == Move)
        {
            double newX = x + stepLength * cos( angle * M_PI / 180.0 );
            double newY = y + stepLength * sin( angle * M_PI / 180.0 );

            x = newX;
            y = newY;
        }
        else if (this->meanings[c] == Push)
        {
            this->states.push(angle);
            this->states.push(y);
            this->states.push(x);
        }
        else if (this->meanings[c] == Pop)
        {
            x = this->states.top();
            this->states.pop();

            y = this->states.top();
            this->states.pop();

            angle = this->states.top();
            this->states.pop();
        }
    }

    file << "</svg>" << std::endl;
    file.close();

    std::cout << "Image successfully generated: " << name << std::endl;
}

/*************************************************
*                                                *
*                   GPU STUFF                    *
*                                                *
**************************************************/

__global__ void countKernel(const char* axiom, int* out, const char* rulesKey, const int* rulesValueLength, int axiomLength, int rulesLength)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    bool found = false;

    if(tid < axiomLength)
    {
        for (int i = 0; i < rulesLength; i++)
        {
            if(axiom[tid] == rulesKey[i])
            {
                out[tid] += rulesValueLength[i];
                found = true;
                break; 
            }
        }

        if(!found)
        {
            out[tid] += 1;
        }
    }
}

__global__ void RewritingKernel(char* input, char* out, char* rulesKey, int* rulesValueLength, char** rulesValue, int* offsetArray, int inputLength, int rulesLength)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    bool found = false;

    if(tid < inputLength)
    {
        for (int i = 0; i < rulesLength; ++i)
        {
            if(input[tid] == rulesKey[i])
            {
                // printf("Input[%d]: %c\tLetter[%d]: %c\n", tid, input[tid], i, rulesKey[i]);

                found = true;

                int size = rulesValueLength[i];
                for (int j = 0; j < size; j++)
                {
                    // printf("tid: %d\tout[%d] = parameters[%d][%d] -> %c\n", tid, offsetArray[tid] + j, i, j, rulesValue[i][j]);
                    out[offsetArray[tid] + j] = rulesValue[i][j];
                }

                break;
            }
        }

        if(!found)
        {
            // printf("out[%d] = input[%d] -> %c\n", offsetArray[tid], tid, input[tid]);
            out[offsetArray[tid]] = input[tid]; 
        }
    }
}

void lsystem::executeOnGPU(const int iteration)
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        std::cerr << "CUDA initialization error: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    this->isGpuUsed = true;
    this->isFreeNeeded = true;

    setupGPUstuff();

    for (int i = 0; i < iteration; i++)
    {
        // std::cout << "Input: " << this->GPUresult << std::endl;
        
        int* valuesLength = count();
        // std::cout << "Counting substitute string length: ";
        // PrintArray(valuesLength, this->GPUresult.length());      

        // std::cout << "Iteration " << i << " string length: " << this->GPUresult.length() << std::endl;
        int* offsetArray;
        hipMallocManaged(&offsetArray, (this->GPUresult.length() + 1) * sizeof(int)); 
        thrust::exclusive_scan(valuesLength, valuesLength + this->GPUresult.length() + 1, offsetArray);

        // std::cout << "Calculating offset array: ";
        // PrintArray(offsetArray, this->GPUresult.length() + 1);

        rewrite(offsetArray);
        // std::cout << "Output: " << this->GPUresult << std::endl;
    
        hipFree(valuesLength);
        hipFree(offsetArray);
    }
}

void lsystem::setupGPUstuff()
{
    this->GPUresult = this->axiom;

    this->rulesLength = this->rules.size();
    hipMallocManaged(&this->rulesKey, this->rulesLength * sizeof(char));
    hipMallocManaged(&this->rulesValueLength, this->rulesLength * sizeof(int));

    // Allocating memory for rules string on GPU
    hipMallocManaged(&this->rulesValue, this->rulesLength * sizeof(char*));    

    int i = 0;
    for (const auto& [key, value] : this->rules)
    {
        this->rulesKey[i] = key;
        this->rulesValueLength[i] = value.length();

        // Allocating on GPU memory arrays of string
        hipMallocManaged(&this->rulesValue[i], (value.length() + 1) * sizeof(char));
        strcpy(this->rulesValue[i], value.c_str());

        i++;
    }
}

int* lsystem::count()
{
    size_t n_character = this->GPUresult.length();

    int threads = n_character > 1024 ? 1024 : n_character;
    int blocks = (n_character + threads - 1) / threads;

    int* out;
    hipMallocManaged(&out, n_character * sizeof(int));
    hipMemset(out, 0, n_character * sizeof(int));

    char* axiom;
    hipMalloc((void**) &axiom, n_character * sizeof(char));
    hipMemcpy(axiom, this->GPUresult.c_str(), n_character, hipMemcpyHostToDevice);

    countKernel<<<blocks, threads>>>(axiom, out, this->rulesKey, this->rulesValueLength, n_character, this->rulesLength);
    hipDeviceSynchronize();

    hipFree(axiom);

    return out;
}

void lsystem::rewrite(int* offsetArray)
{
    size_t n_character = this->GPUresult.length();

    char* input;
    hipMalloc((void**) &input, n_character * sizeof(char));
    hipMemcpy(input, this->GPUresult.c_str(), n_character * sizeof(char), hipMemcpyHostToDevice);

    char* output;
    hipMalloc((void**) &output, offsetArray[n_character] * sizeof(char));

    int threads = n_character > 1024 ? 1024 : n_character;
    int blocks = (n_character + threads - 1) / threads;

    RewritingKernel<<<blocks, threads>>>(input, output, this->rulesKey, this->rulesValueLength, this->rulesValue, offsetArray, n_character, this->rulesLength);
    hipDeviceSynchronize();

    char* out = (char*) malloc(offsetArray[n_character] * sizeof(char));
    hipMemcpy(out, output, offsetArray[n_character] * sizeof(char), hipMemcpyDeviceToHost);

    this->GPUresult = out;

    hipFree(input);
    hipFree(output);
    free(out);
}
#include "hip/hip_runtime.h"
#include "lsystem.h"

lsystem::lsystem(std::string axiom, std::map<char, std::string> rules): axiom(""), rules({})
{
   this->axiom = axiom;
   this->rules = rules; 

    for (const auto& pair : this->rules) {
        const std::string& value = pair.second;

        if(this->longestRule < value.length())
        {
            this->longestRule = value.length();
        }
    }
}

lsystem::lsystem(std::string axiom, std::string rules): axiom(""), rules({})
{
    this->axiom = axiom;

    try
    {
        parseString(rules);
    }
    catch(const RulesWrongFormatException &e)
    {
        std::cout << "Exception occured: " << e.what() << std::endl;
    }
    
}

lsystem::lsystem(const lsystem& other)
{
    this->axiom = other.axiom;
    this->rules = other.rules;
}

lsystem& lsystem::operator=(const lsystem &other)
{
    if(this != &other)
    {
        lsystem tmp(other);
        std::swap(tmp.axiom, this->axiom);
        std::swap(tmp.rules, this->rules);
    }

    return *this;
}

lsystem::~lsystem()
{

}

std::string lsystem::get_axiom() const
{
    return this->axiom;
}

std::map<char, std::string> lsystem::get_rules() const
{
    return this->rules;
}

std::string lsystem::get_result() const
{
    return this->result;
}

std::ostream &operator<<(std::ostream &os, const lsystem &system)
{
    if(system.rules.size() == 0 || system.axiom.empty())
    {
        throw RulesEmptyException();
    }

    os << "l-system with axiom: " << system.get_axiom() << " and rules: ";

    for (const auto& [key, value] : system.get_rules())
    {
        os << key << " → " << value << "    ";
    }

    return os;
}

void lsystem::parseString(const std::string rules)
{
    std::stringstream ss(rules);
    char key;
    std::string value;

    while(ss >> key)
    {
        if(!(ss >> value))
        {
            this->rules = {};
            this->axiom = "";
            throw RulesWrongFormatException();
        }

        // std::cout << "Key: " << key << " value: " << value << std::endl;

        if(this->longestRule < value.length())
        {
            this->longestRule = value.length();
        }

        this->rules[key] = value;
    }
}

void lsystem::setCustomMeaning(std::string symbols, int meaning)
{
    for(char c: symbols)
    {
        if(c != ' ')
        {
            this->meanings[c] = meaning;
        }
    }
}

void lsystem::execute(const int iteration)
{
    std::string result = this->axiom;

    for(int i = 0; i < iteration; i++)
    {
        std::string tmp = "";

        for(char c: result)
        {
            if(this->rules.find(c) != this->rules.end())
            {
                tmp += this->rules[c];
            }
            else
            {
                tmp += c;
            }
        }

        result = tmp;
    }

    // std::cout << "The result is: " << result << std::endl;
    this->result = result;
}

void lsystem::execute(const int iteration, const bool useGPU)
{
    // int nKeys = this->rules.size();

    // char keys[nKeys];
    // const char* values[nKeys];

    // int i = 0;
    // for (auto it = this->rules.begin(); it != this->rules.end(); ++it) {
    //     keys[i] = it -> first;
    //     values[i] = this->rules[keys[i]].c_str();
    //     i++;
    // }

    // size_t estimatedOutputsize = this->axiom.length() * this->longestRule * iteration;

}

__global__ void lsystemKernel(char* input, char* output, int inputLength, int outputLength, int iterations)
{

}

void lsystem::write(std::string name) const
{
    std::ofstream file("../Results/" + name + ".txt", std::ios::out);
    
    if (file.is_open())
    {
        file << this->result;
        file.close();

        std::cout << "The string has been written to the file." << std::endl;
    }
    else
    {
        std::cout << "Unable to open file." << std::endl;
    }
}

void lsystem::draw(const std::string name, const double turnAngle, const int stepLength, const int startingDirection)
{
    if(this->meanings.size() == 0)
    {
        this->meanings = this->defaultMeaning;
    }

    std::ofstream file("../Results/" + name + ".svg");
    if (!file.is_open()) {
        std::cout << "Errore nell'apertura del file." << std::endl;
        return;
    }

    file << "<svg xmlns=\"http://www.w3.org/2000/svg\" version=\"1.1\">" << std::endl;

    double x = 0.0;
    double y = 0.0;

    double angle = startingDirection;

    for (char c : this->result) {
        if (this->meanings[c] == DRAW) {
            
            double newX = x + stepLength * cos( angle * M_PI / 180.0 );
            double newY = y + stepLength * sin( angle * M_PI / 180.0 );

            file << "<line x1=\"" << static_cast<int>(x) << "\" y1=\"" << static_cast<int>(y)<< "\" x2=\"" << static_cast<int>(newX) << "\" y2=\"" << static_cast<int>(newY) << "\" stroke=\"black\" stroke-width=\"0.8\"/>" << std::endl;

            x = newX;
            y = newY;
        }
        else if (this->meanings[c] == TURNRIGHT) {
            angle += turnAngle;
        }
        else if (this->meanings[c] == TURNLEFT) {
            angle -= turnAngle;
        }
        else if(this->meanings[c] == MOVE)
        {
            double newX = x + stepLength * cos( angle * M_PI / 180.0 );
            double newY = y + stepLength * sin( angle * M_PI / 180.0 );

            x = newX;
            y = newY;
        }
        else if (this->meanings[c] == PUSH)
        {
            this->positions.push(std::make_pair(x, y));
            this->orientations.push(angle);
        }
        else if (this->meanings[c] == POP)
        {
            x = this->positions.top().first;
            y = this->positions.top().second;
            angle = this->orientations.top();

            this->positions.pop();
            this->orientations.pop();
        }
    }

    file << "</svg>" << std::endl;
    file.close();

    std::cout << "Image successfully generated: " << name << std::endl;
}
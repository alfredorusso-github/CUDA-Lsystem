#include "hip/hip_runtime.h"
#include "lsystem.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include<math.h>

lsystem::lsystem(std::string axiom, std::string rules): axiom(""), rules({})
{
    this->axiom = axiom;

    if(axiom.length() == 0)
    {
        throw AxiomEmptyExecption();
    }

    if(rules.length() == 0)
    {
        throw RulesEmptyException();
    }

    try
    {
        parseString(rules);
    }
    catch(const RulesWrongFormatException &e)
    {
        std::cout << "Exception occured: " << e.what() << std::endl;
    }
    
}

lsystem::lsystem(std::string axiom, std::map<char, std::string> rules): axiom(""), rules({})
{
    if(axiom.length() == 0)
    {
        throw AxiomEmptyExecption();
    } 

    if(rules.empty())
    {
        throw RulesEmptyException();
    }  

    this->axiom = axiom;
    this->rules = rules;
}

lsystem::lsystem(const lsystem& other)
{
    this->axiom = other.axiom;
    this->rules = other.rules;
}

lsystem& lsystem::operator=(const lsystem &other)
{
    if(this != &other)
    {
        lsystem tmp(other);
        std::swap(tmp.axiom, this->axiom);
        std::swap(tmp.rules, this->rules);
    }

    return *this;
}

lsystem::~lsystem()
{
    freeMemory();
}

void lsystem::freeMemory()
{
    if(this->isGpuUsed && this->isFreeNeeded)
    {
        hipFree(this->rulesKey);
        hipFree(this->rulesValueLength);

        for (int i = 0; i < this->rulesLength; i++)
        {
            hipFree(this->rulesValue[i]); 
        }
        hipFree(this->rulesValue);

        isFreeNeeded = false;
        isGpuUsed = false;
    }
}

std::string lsystem::get_axiom() const
{
    return this->axiom;
}

std::map<char, std::string> lsystem::get_rules() const
{
    return this->rules;
}

std::string lsystem::get_result() const
{
    if(this->result.length() == 0)
    {
        throw EmptyResultExecption();
    }

    return this->result;
}

std::string lsystem::get_GPUResult() const
{
    if(this->GPUresult.length() == 0)
    {
        throw EmptyGpuResultExecption();
    }

    return this->GPUresult;
}

std::map<char, int> lsystem::get_meanings() const
{
    return this->meanings;
}

std::ostream &operator<<(std::ostream &os, const lsystem &system)
{
    if(system.rules.size() == 0 || system.axiom.empty())
    {
        throw RulesEmptyException();
    }

    os << "l-system specs" << std::endl;
    os << "axiom: " << system.get_axiom() << std::endl;

    os << "rules: "; 
    for (const auto& [key, value] : system.get_rules())
    {
        os << key << " → " << value << "  ";
    }

    os << std::endl <<  "meanings: ";
    for (const auto& [key, value] : system.get_meanings())
    {
        os << key << " → " << system.symbolMeaningsName[value] << "  ";
    }

    return os;
}

void lsystem::parseString(const std::string rules)
{
    std::stringstream ss(rules);
    char key;
    std::string value;

    while(ss >> key)
    {
        if(!(ss >> value))
        {
            this->rules = {};
            this->axiom = "";
            throw RulesWrongFormatException();
        }

        // std::cout << "Key: " << key << " value: " << value << std::endl;

        this->rules[key] = value;
    }
}

void lsystem::setMeaning(std::string symbols, int meaning)
{
    for(char c: symbols)
    {
        if(c != ' ')
        {
            this->meanings[c] = meaning;
        }
    }
}

void lsystem::execute(const int iteration)
{
    std::string result = this->axiom;

    for(int i = 0; i < iteration; i++)
    {
        std::string tmp = "";

        for(char c: result)
        {
            if(this->rules.find(c) != this->rules.end())
            {
                tmp += this->rules[c];
            }
            else
            {
                tmp += c;
            }
        }

        result = tmp;
    }

    // std::cout << "The result is: " << result << std::endl;
    this->result = result;
}

void lsystem::write(const std::string name, const bool writeGPUResult) const
{
    if(writeGPUResult && this->GPUresult.length() == 0)
    {
        throw EmptyGpuResultExecption();
    }

    if(!writeGPUResult && this->result.length() == 0)
    {
        throw EmptyResultExecption();
    }

    std::ofstream file("../Results/" + name + ".txt", std::ios::out);

    std::string whatToWrite = writeGPUResult ? this->GPUresult : this->result;
    
    if (file.is_open())
    {
        file << this->result;
        file.close();

        std::cout << "The string has been written to the file." << std::endl;
    }
    else
    {
        std::cout << "Unable to open file." << std::endl;
    }
}

void lsystem::draw(const std::string name, const double turnAngle, const int stepLength, const bool drawGPUResult, const int startingDirection)
{
    if(drawGPUResult && this->GPUresult.length() == 0)
    {
        throw EmptyGpuResultExecption();
    }

    if(!drawGPUResult && this->result.length() == 0)
    {
        throw EmptyResultExecption();
    }

    std::ofstream file("../Results/" + name + ".svg");
    if (!file.is_open()) {
        std::cerr << "An error occured when tried to open/create the files." << std::endl;
        return;
    }

    file << "<svg xmlns=\"http://www.w3.org/2000/svg\" version=\"1.1\">" << std::endl;

    double x = 0.0;
    double y = 0.0;

    double angle = startingDirection;

    std::string whatToDraw = drawGPUResult ? this->GPUresult : this->result;

    for (char c : whatToDraw) {
        if (this->meanings[c] == Draw) {
            
            double newX = x + stepLength * cos( angle * M_PI / 180.0 );
            double newY = y + stepLength * sin( angle * M_PI / 180.0 );

            file << "<line x1=\"" << static_cast<int>(x) << "\" y1=\"" << static_cast<int>(y)<< "\" x2=\"" << static_cast<int>(newX) << "\" y2=\"" << static_cast<int>(newY) << "\" stroke=\"black\" stroke-width=\"0.8\"/>" << std::endl;

            x = newX;
            y = newY;
        }
        else if (this->meanings[c] == Turnright) {
            angle += turnAngle;
        }
        else if (this->meanings[c] == Turnleft) {
            angle -= turnAngle;
        }
        else if(this->meanings[c] == Move)
        {
            double newX = x + stepLength * cos( angle * M_PI / 180.0 );
            double newY = y + stepLength * sin( angle * M_PI / 180.0 );

            x = newX;
            y = newY;
        }
        else if (this->meanings[c] == Push)
        {
            this->states.push(angle);
            this->states.push(y);
            this->states.push(x);
        }
        else if (this->meanings[c] == Pop)
        {
            x = this->states.top();
            this->states.pop();

            y = this->states.top();
            this->states.pop();

            angle = this->states.top();
            this->states.pop();
        }
    }

    file << "</svg>" << std::endl;
    file.close();

    std::cout << "Image successfully generated: " << name << std::endl;
}

/*************************************************
*                                                *
*                   GPU STUFF                    *
*                                                *
**************************************************/

__global__ void countKernel(const char* axiom, int* out, const char* rulesKey, const int* rulesValueLength, int axiomLength, int rulesLength)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    bool found = false;

    if(tid < axiomLength)
    {
        for (int i = 0; i < rulesLength; i++)
        {
            if(axiom[tid] == rulesKey[i])
            {
                out[tid] += rulesValueLength[i];
                found = true;
                break; 
            }
        }

        if(!found)
        {
            out[tid] += 1;
        }
    }
}

__global__ void countKernelWithModules(const char* axiom, int* out, const char* rulesKey, const int* rulesValueLength, int axiomLength, int rulesLength, int n_modules, int modulesLength)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    bool found = false;

    if(tid < n_modules)
    {
        for (int i = 0; i < modulesLength; i++)
        {
            if(tid * modulesLength + i == axiomLength) break;

            for (int j = 0; j < rulesLength; j++)
            {
                if(axiom[tid * modulesLength + i] == rulesKey[j])
                {
                    out[tid] += rulesValueLength[j];
                    found = true;
                }
            }

            if(!found)
            {
                out[tid] += 1;
            }

            found = false;
        }
    }
}

__global__ void RewritingKernel(char* input, char* out, char* rulesKey, int* rulesValueLength, char** rulesValue, int* offsetArray, int inputLength, int rulesLength)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    bool found = false;

    if(tid < inputLength)
    {
        for (int i = 0; i < rulesLength; i++)
        {
            if(input[tid] == rulesKey[i])
            {
                // printf("Input[%d]: %c\tLetter[%d]: %c\n", tid, input[tid], i, rulesKey[i]);

                found = true;

                int size = rulesValueLength[i];
                for (int j = 0; j < size; j++)
                {
                    // printf("tid: %d\tout[%d] = parameters[%d][%d] -> %c\n", tid, offsetArray[tid] + j, i, j, rulesValue[i][j]);
                    out[offsetArray[tid] + j] = rulesValue[i][j];
                }

                break;
            }
        }

        if(!found)
        {
            // printf("out[%d] = input[%d] -> %c\n", offsetArray[tid], tid, input[tid]);
            printf("");
            out[offsetArray[tid]] = input[tid]; 
        }
    }
}

__global__ void RewritingKernelWithModules(char* input, char* out, char* rulesKey, int* rulesValueLength, char** rulesValue, int* offsetArray, int inputLength, int rulesLength, int n_modules, int modulesLength)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    bool found = false;

    if(tid < n_modules)
    {
        int index = 0;

        for (int i = 0; i < modulesLength; i++)
        {
            if(tid * modulesLength + i == inputLength) break;

            // printf("tid: %d \t input[%d] = %c \n", tid, tid * modulesLength + i, input[tid * modulesLength + i]);
            
            for (int j = 0; j < rulesLength; j++)
            {
                if(input[tid * modulesLength + i] == rulesKey[j])
                {
                    found = true;
                    // if (tid == 6) printf("input[%d] = %c found at rulesKey[%d] = %c\n", tid * modulesLength + i, input[tid * modulesLength + i], j, rulesKey[j]);

                    int size = rulesValueLength[j];
                    for (int k = 0; k < size; k++)
                    {
                        out[offsetArray[tid] + index] = rulesValue[j][k];
                        // printf("out[%d] = %c\n", offsetArray[tid] + index, rulesValue[j][k]);
                        index += 1;
                    }
                }
            }

            if(!found)
            {
                out[offsetArray[tid] + index] = input[tid * modulesLength + i];
                // printf("Not found, out[%d] = %c\n", offsetArray[tid] + index, input[tid * modulesLength + i]);
                index += 1;
            }

            found = false;
        }
    }
}

void lsystem::executeOnGPU(const int iteration)
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        std::cerr << "CUDA initialization error: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    this->isGpuUsed = true;
    this->isFreeNeeded = true;

    setupGPUstuff();

    for (int i = 0; i < iteration; i++)
    {
        // std::cout << "Input: " << this->GPUresult << std::endl;
        
        int* valuesLength = count();
        // std::cout << "Counting substitute string length: ";
        // PrintArray(valuesLength, this->GPUresult.length());      

        // std::cout << "Iteration " << i << " string length: " << this->GPUresult.length() << std::endl;
        int* offsetArray;
        hipMallocManaged(&offsetArray, (this->GPUresult.length() + 1) * sizeof(int)); 
        thrust::exclusive_scan(valuesLength, valuesLength + this->GPUresult.length() + 1, offsetArray);

        // for (int i = 0; i < this->GPUresult.length() + 1; i++)
        // {
        //     printf("%d ", offsetArray[i]);
        // }
        // printf("\n");        

        // std::cout << "Calculating offset array: ";
        // PrintArray(offsetArray, this->GPUresult.length() + 1);

        rewrite(offsetArray);
        // std::cout << "Output: " << this->GPUresult << std::endl;
    
        hipFree(valuesLength);
        hipFree(offsetArray);
    }
}

void lsystem::executeOnGPUWithModules(const int iteration, const int modulesLength)
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        std::cerr << "CUDA initialization error: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    this->isGpuUsed = true;
    this->isFreeNeeded = true;

    setupGPUstuff();

    for (int i = 0; i < iteration; i++)
    {
        int* valuesLength = countWithModules(modulesLength);    

        int* offsetArray;
        int n_modules = ceil(static_cast<float>(this->GPUresult.length()) / modulesLength);
        hipMallocManaged(&offsetArray, (n_modules + 1) * sizeof(int));
        thrust::exclusive_scan(valuesLength, valuesLength + n_modules + 1, offsetArray);

        rewriteWithModules(offsetArray, modulesLength);          

        hipFree(valuesLength);
        hipFree(offsetArray);
    }
}

void lsystem::setupGPUstuff()
{
    this->GPUresult = this->axiom;

    this->rulesLength = this->rules.size();
    hipMallocManaged(&this->rulesKey, this->rulesLength * sizeof(char));
    hipMallocManaged(&this->rulesValueLength, this->rulesLength * sizeof(int));

    // Allocating memory for rules string on GPU
    hipMallocManaged(&this->rulesValue, this->rulesLength * sizeof(char*));    

    int i = 0;
    for (const auto& [key, value] : this->rules)
    {
        this->rulesKey[i] = key;
        this->rulesValueLength[i] = value.length();

        // Allocating on GPU memory arrays of string
        hipMallocManaged(&this->rulesValue[i], (value.length() + 1) * sizeof(char));
        strcpy(this->rulesValue[i], value.c_str());

        i++;
    }
}

int* lsystem::count()
{
    size_t n_character = this->GPUresult.length();

    int threads = n_character > 1024 ? 1024 : n_character;
    int blocks = (n_character + threads - 1) / threads;

    int* out;
    hipMallocManaged(&out, n_character * sizeof(int));
    hipMemset(out, 0, n_character * sizeof(int));

    char* axiom;
    hipMalloc((void**) &axiom, n_character * sizeof(char));
    hipMemcpy(axiom, this->GPUresult.c_str(), n_character, hipMemcpyHostToDevice);

    countKernel<<<blocks, threads>>>(axiom, out, this->rulesKey, this->rulesValueLength, n_character, this->rulesLength);
    hipDeviceSynchronize();

    hipFree(axiom);

    return out;
}

int* lsystem::countWithModules(const int modulesLength)
{
    size_t n_character = this->GPUresult.length();
    size_t n_modules = ceil(static_cast<float>(this->GPUresult.length()) / modulesLength);
    // std::cout << "Modules: " << n_modules << std::endl;

    int threads = n_modules >  1024 ? 1024 : n_modules;
    int blocks = (n_modules + threads - 1) / threads;

    int* out;
    hipMallocManaged(&out, n_modules * sizeof(int));
    hipMemset(out, 0, n_modules * sizeof(int));

    char* axiom;
    hipMalloc((void**) &axiom, n_character * sizeof(char));
    hipMemcpy(axiom, this->GPUresult.c_str(), n_character, hipMemcpyHostToDevice);

    countKernelWithModules<<<blocks, threads>>>(axiom, out, this->rulesKey, this->rulesValueLength, n_character, this->rulesLength, n_modules, modulesLength); 
    hipDeviceSynchronize();

    hipFree(axiom);
    
    return out;
}

void lsystem::rewrite(int* offsetArray)
{
    size_t n_character = this->GPUresult.length();

    char* input;
    hipMalloc((void**) &input, n_character * sizeof(char));
    hipMemcpy(input, this->GPUresult.c_str(), n_character * sizeof(char), hipMemcpyHostToDevice);

    char* output;
    hipMalloc((void**) &output, offsetArray[n_character] * sizeof(char));

    int threads = n_character > 1024 ? 1024 : n_character;
    int blocks = (n_character + threads - 1) / threads;

    RewritingKernel<<<blocks, threads>>>(input, output, this->rulesKey, this->rulesValueLength, this->rulesValue, offsetArray, n_character, this->rulesLength);
    hipDeviceSynchronize();

    char* out = (char*) malloc(offsetArray[n_character] * sizeof(char));
    hipMemcpy(out, output, offsetArray[n_character] * sizeof(char), hipMemcpyDeviceToHost);

    this->GPUresult = out;

    hipFree(input);
    hipFree(output);
    free(out);
}

void lsystem::rewriteWithModules(int* offsetArray, const int modulesLength)
{
    size_t n_character = this->GPUresult.length();
    size_t n_modules = ceil(static_cast<float>(this->GPUresult.length()) / modulesLength);
    size_t output_characters = offsetArray[n_modules];

    char* input;
    hipMalloc((void**) &input, n_character * sizeof(char));
    hipMemcpy(input, this->GPUresult.c_str(), n_character * sizeof(char), hipMemcpyHostToDevice);

    char* output = nullptr;
    hipMalloc((void**) &output, output_characters * sizeof(char));
    hipMemset(output, 'a', output_characters * sizeof(char));

    int threads = n_modules > 1024 ? 1024 : n_modules;
    int blocks = (n_modules + threads - 1) / threads;

    RewritingKernelWithModules<<<blocks, threads>>>(input, output, this->rulesKey, this->rulesValueLength, this->rulesValue, offsetArray, n_character, this->rulesLength, n_modules, modulesLength);
    hipDeviceSynchronize();

    char* out = (char*) malloc(output_characters * sizeof(char));
    hipMemcpy(out, output, output_characters * sizeof(char), hipMemcpyDeviceToHost);

    this->GPUresult = out;

    hipFree(input);
    hipFree(output);
    free(out);
}